#include "hip/hip_runtime.h"
#include <stdio.h>
#include <array>
#include <numeric>
#include <iostream>
#include <iterator>
#include <memory>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"

#include "Matrix.h"
#include "CudaHelpers.h"

const unsigned int TILE_SIZE_X = 16;
const unsigned int TILE_SIZE_Y = TILE_SIZE_X;
const unsigned int DIM_X = 1024;
const unsigned int DIM_Y = DIM_X;
const unsigned int NUM_REPS = 100;

template <typename T>
__global__ void baseLineCopy(const T * const in, T * const out, unsigned int width, unsigned int height) {
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;

	out[idxIn] = in[idxIn];
}

template <typename T>
__global__ void baseLineCopyShared(const T * const in, T * const out, unsigned int width, unsigned int height) {
	__shared__ T cache[TILE_SIZE_Y][TILE_SIZE_X];
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;

	cache[threadIdx.y][threadIdx.x] = in[idxIn];
	__syncthreads();

	out[idxIn] = cache[threadIdx.y][threadIdx.x];
}

template <typename T>
__global__ void naiveTranspose(const T * const in, T * const out, unsigned int width, unsigned int height) {
	for (unsigned int j = 0; j < height; j++) {
		for (unsigned int i = 0; i < width; i++) {
			auto idxIn = j * width + i;
			auto idxOut = i * height + j;
			out[idxOut] = in[idxIn];
		}
	}
}

template <typename T>
__global__ void naiveParallelTranspose(const T * const in, T * const out, unsigned int width, unsigned int height) {
	unsigned int idxIn = blockIdx.x * width + threadIdx.x;
	unsigned int idxOut = threadIdx.x * height + blockIdx.x;

	out[idxOut] = in[idxIn];
}

template <typename T>
__global__ void naiveBlockWiseParallelTranspose(const T * const in, T * const out, unsigned int width, unsigned int height) {
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;
	unsigned int idxOut = xIndex * height + yIndex;

	out[idxOut] = in[idxIn];
}

template <typename T>
__global__ void naiveSharedBlockWiseParallelTranspose(const T * const in, T * const out, unsigned int width, unsigned int height) {
	__shared__ T cache[TILE_SIZE_Y][TILE_SIZE_X];
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;
	unsigned int idxOut = xIndex * height + yIndex;

	cache[threadIdx.y][threadIdx.x] = in[idxIn];
	__syncthreads();

	out[idxOut] = cache[threadIdx.y][threadIdx.x];
}

template <typename T>
__global__ void coalescedSharedBlockWiseParallelTranspose(const T * const in, T * const out, unsigned int width, unsigned int height) {
	__shared__ T cache[TILE_SIZE_Y][TILE_SIZE_X];
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;

	// write out consecutively and do transposing by swapping x and y when reading from shared memory
	unsigned int xIndexOut = blockIdx.y * blockDim.y + threadIdx.x;
	unsigned int yIndexOut = blockIdx.x * blockDim.x + threadIdx.y;
	unsigned int idxOut = yIndexOut * height + xIndexOut;

	cache[threadIdx.y][threadIdx.x] = in[idxIn];
	__syncthreads();

	out[idxOut] = cache[threadIdx.x][threadIdx.y];
}

template <typename T>
__global__ void coalescedSharedBlockWiseParallelTransposeNoBankConflicts(const T * const in, T * const out, unsigned int width, unsigned int height) {
	__shared__ T cache[TILE_SIZE_Y][TILE_SIZE_X + 1];
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idxIn = yIndex * width + xIndex;

	// write out consecutively and do transposing by swapping x and y when reading from shared memory
	unsigned int xIndexOut = blockIdx.y * blockDim.y + threadIdx.x;
	unsigned int yIndexOut = blockIdx.x * blockDim.x + threadIdx.y;
	unsigned int idxOut = yIndexOut * height + xIndexOut;

	cache[threadIdx.y][threadIdx.x] = in[idxIn];
	__syncthreads();

	out[idxOut] = cache[threadIdx.x][threadIdx.y];
}

template <typename T, unsigned int WIDTH, unsigned int HEIGHT>
void transposeCpu(const Matrix<T, WIDTH, HEIGHT>& in, Matrix<T, HEIGHT, WIDTH>& out) {
	for (unsigned int j = 0; j < HEIGHT; j++) {
		for (unsigned int i = 0; i < WIDTH; i++) {
			auto inIndex = j * WIDTH + i;
			auto outIndex = i * HEIGHT + j;
			out[outIndex] = in[inIndex];
		}
	}
}

template<typename T>
struct KernelParams {
	const T * const d_in;
	T * const d_out;
	unsigned int width;
	unsigned int height;
	dim3 gridDim;
	dim3 blockDim;
};

template <typename T>
float averageTime(void(*kernel)(const T * const, T * const, unsigned int, unsigned int),
				  const T * const in, T * const out, unsigned int width, unsigned int height,
				  dim3 grid, dim3 block, unsigned int numReps) {
	float kernelTime = -1.0f;
	hipEvent_t start;
	hipEvent_t stop;
	try {
		checkCudaStatus(hipEventCreate(&start));
		checkCudaStatus(hipEventCreate(&stop));
		// Clear error status
		checkCudaStatus(hipGetLastError());

		// warmup to avoid timing startup
		kernel << <grid, block >> > (in, out, width, height);

		checkCudaStatus(hipEventRecord(start, 0));

		for (unsigned int i = 0; i < numReps; i++)
		{
			kernel << <grid, block >> > (in, out, width, height);
			// Ensure no launch failure
			checkCudaStatus(hipGetLastError());
		}

		checkCudaStatus(hipEventRecord(stop, 0));
		checkCudaStatus(hipEventSynchronize(stop));
		checkCudaStatus(hipEventElapsedTime(&kernelTime, start, stop));
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	catch (CudaException e) {
		std::cerr << "Kernel launch failed: " << e.what() << std::endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	return kernelTime / numReps;
}

template <typename T>
float averageTime(void(*kernel)(const T * const, T * const, unsigned int, unsigned int),
				  const KernelParams<T>& params, unsigned int numReps) {
	float kernelTime = -1.0f;
	hipEvent_t start;
	hipEvent_t stop;
	try {
		checkCudaStatus(hipEventCreate(&start));
		checkCudaStatus(hipEventCreate(&stop));
		// Clear error status
		checkCudaStatus(hipGetLastError());

		// warmup to avoid timing startup
		kernel << <params.gridDim, params.blockDim >> > (params.d_in, params.d_out, params.width, params.height);

		checkCudaStatus(hipEventRecord(start, 0));

		for (unsigned int i = 0; i < numReps; i++)
		{
			kernel << <params.gridDim, params.blockDim >> > (params.d_in, params.d_out, params.width, params.height);
			// Ensure no launch failure
			checkCudaStatus(hipGetLastError());
		}

		checkCudaStatus(hipEventRecord(stop, 0));
		checkCudaStatus(hipEventSynchronize(stop));
		checkCudaStatus(hipEventElapsedTime(&kernelTime, start, stop));
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	catch (CudaException e) {
		std::cerr << "Kernel launch failed: " << e.what() << std::endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	return kernelTime / numReps;
}

void printStatistics(float avgTime, size_t mem_size) {
	static const unsigned int BYTES_PER_GIGABYTE = 1024 * 1024 * 1024;
	float kernelBandwidth = 2.0f * 1000.0f * mem_size / (BYTES_PER_GIGABYTE) / (avgTime);
	std::cout << avgTime << " " << mem_size << " " << kernelBandwidth << std::endl;
}

template <typename T, unsigned int WIDTH, unsigned int HEIGHT>
void compare(const Matrix<T, HEIGHT, WIDTH>& toCompare, const Matrix<T, HEIGHT, WIDTH>& groundTruth) {
	std::cout << (toCompare == groundTruth ? "Test Passed" : "Test Failed") << std::endl;
}

template <typename T>
void testKernel(void(*kernel)(const T * const, T * const, unsigned int, unsigned int),
				const KernelParams<T>& params, unsigned int numReps) {
	float avgTime = averageTime(kernel, params, numReps);
	printStatistics(avgTime, params.width * params.height * sizeof(T));
}

template <typename T, unsigned int WIDTH, unsigned int HEIGHT >
void testKernel(void(*kernel)(const T * const, T * const, unsigned int, unsigned int),
				const KernelParams<T>& params, unsigned int numReps,
				Matrix<T, WIDTH, HEIGHT>& out, const Matrix<T, WIDTH, HEIGHT>& groundTruth) {
	testKernel(kernel, params, numReps);
	hipMemcpy(out.getData(), params.d_out, WIDTH * HEIGHT * sizeof(T), hipMemcpyDefault);
	compare(out, groundTruth);
}

template <typename T, unsigned int WIDTH, unsigned int HEIGHT>
void transposeGpu(const Matrix<T, WIDTH, HEIGHT>& in, Matrix<T, HEIGHT, WIDTH>& out, const Matrix<T, HEIGHT, WIDTH>& groundTruth) {
	hipSetDevice(0);
	T* d_in;
	T* d_out;
	const size_t byteSize = WIDTH * HEIGHT * sizeof(T);
	hipMalloc(&d_in, byteSize);
	hipMalloc(&d_out, byteSize);
	hipMemcpy(d_in, in.getData(), byteSize, hipMemcpyDefault);

	dim3 gridTiled(WIDTH / TILE_SIZE_X, HEIGHT / TILE_SIZE_Y);
	dim3 blockTiled(TILE_SIZE_X, TILE_SIZE_Y);

	KernelParams<T> kernelParams = { d_in, d_out, WIDTH, HEIGHT, gridTiled, blockTiled };

	testKernel(baseLineCopy, kernelParams, NUM_REPS);
	testKernel(baseLineCopyShared, kernelParams, NUM_REPS);

	if (WIDTH * HEIGHT < 4096) {
		dim3 gridOne(1);
		dim3 blockOne(1);
		kernelParams.gridDim = gridOne;
		kernelParams.blockDim = blockOne;
		testKernel(naiveTranspose, kernelParams, NUM_REPS, out, groundTruth);
	}

	dim3 gridRowWise(HEIGHT);
	dim3 blockRowWise(WIDTH);
	kernelParams.gridDim = gridRowWise;
	kernelParams.blockDim = blockRowWise;
	testKernel(naiveParallelTranspose, kernelParams, NUM_REPS, out, groundTruth);

	kernelParams.gridDim = gridTiled;
	kernelParams.blockDim = blockTiled;
	testKernel(naiveBlockWiseParallelTranspose, kernelParams, NUM_REPS, out, groundTruth);
	testKernel(naiveSharedBlockWiseParallelTranspose, kernelParams, NUM_REPS, out, groundTruth);
	testKernel(coalescedSharedBlockWiseParallelTranspose, kernelParams, NUM_REPS, out, groundTruth);
	testKernel(coalescedSharedBlockWiseParallelTransposeNoBankConflicts, kernelParams, NUM_REPS, out, groundTruth);

	hipFree(d_in);
	hipFree(d_out);

	hipDeviceReset();
}

int main() {
	Matrix<int, DIM_X, DIM_Y> matrix;
	std::iota(&matrix[0], &matrix[DIM_X * DIM_Y], 0);

	//std::cout << matrix << std::endl;

	Matrix<int, DIM_Y, DIM_X> groundTruth;
	transposeCpu(matrix, groundTruth);
	//std::cout << groundTruth << std::endl;

	Matrix<int, DIM_Y, DIM_X> gpuTransposed;
	transposeGpu(matrix, gpuTransposed, groundTruth);
	//std::cout << gpuTransposed << std::endl;

	return 0;
}